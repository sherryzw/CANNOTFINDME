#include "hip/hip_runtime.h"
//Salted MD5 brute force with CUDA
//By FireXware, Aug 2nd 2010.
//OSSBox.com

//TODO: rename variables so they are called length, not max, max means max size, length means length including null terimnation
//TODO: optimize
//TODO: get command line arguments
//TODO: md5 2nd block

#define MAX_BRUTE_LENGTH 14 
#define MAX_SALT_LENGTH 38
#define MAX_TOTAL (MAX_SALT_LENGTH + MAX_BRUTE_LENGTH + MAX_SALT_LENGTH)

//Performance:
#define BLOCKS 64
#define THREADS_PER_BLOCK 128
#define MD5_PER_KERNEL 200
#define OUTPUT_INTERVAL 20

__device__ __constant__ unsigned char cudaBrute[MAX_BRUTE_LENGTH];
__device__ __constant__ unsigned char cudaLeftSalt[MAX_SALT_LENGTH];
__device__ __constant__ unsigned char cudaRightSalt[MAX_SALT_LENGTH];
__device__ __constant__ unsigned char cudaCharSet[95];
__device__ unsigned char correctPass[MAX_TOTAL];

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include "md5.cu" //This contains our MD5 helper functions
#include "md5kernel.cu" //the CUDA thread

void checkCUDAError(const char *msg);

void ZeroFill(unsigned char* toFill, int length)
{
	int i = 0; 
	for(i = 0; i < length; i++)
		toFill[i] = 0;
}

bool BruteIncrement(unsigned char* brute, int setLen, int wordLength, int incrementBy)
{
	int i = 0;
	while(incrementBy > 0 && i < wordLength)
	{
		int add = incrementBy + brute[i];
		brute[i] = add % setLen;
		incrementBy = add / setLen;
		i++;
	}
	
	return incrementBy != 0; //we are done if there is a remainder, because we have looped over the max
}

int main( int argc,char* argv[]) 
{
	int wordLength = 6;
	int charSetLen = 0;


	int numThreads = BLOCKS * THREADS_PER_BLOCK;

	unsigned char currentBrute[MAX_BRUTE_LENGTH];
	unsigned char leftSalt[MAX_SALT_LENGTH];
	unsigned char rightSalt[MAX_SALT_LENGTH];

	unsigned char cpuCorrectPass[MAX_TOTAL];

	ZeroFill(currentBrute, MAX_BRUTE_LENGTH);
	ZeroFill(cpuCorrectPass, MAX_TOTAL);
	ZeroFill(leftSalt, MAX_SALT_LENGTH);
	ZeroFill(rightSalt, MAX_SALT_LENGTH);

	//for this example, we will crack the hash of "http://ossbox.com"
	//we will use "http://" as the salt on the left and ".com" as the salt on the right
	//so our code has to brute force 'ossbox'
	charSetLen = 26;
	unsigned char charSet[charSetLen];
	memcpy(charSet, "abcdefghijklmnopqrstuvwxyz", charSetLen);

	unsigned char hash[32];

	memcpy(hash, argv[1], 32);
	//memcpy(hash, "88461973aeb1d7475dbf0de499c963d9", 32);
	
	

	memcpy(leftSalt, "http://", 7); 
	memcpy(rightSalt, ".com", 4);
	
	//turn the correct hash into it's four parts
	uint v1, v2, v3, v4;
	md5_to_ints(hash,&v1,&v2,&v3,&v4);

	//copy the salts to global
	hipMemcpyToSymbol(HIP_SYMBOL(cudaLeftSalt), &leftSalt, MAX_SALT_LENGTH, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(cudaRightSalt), &rightSalt, MAX_SALT_LENGTH, 0, hipMemcpyHostToDevice);

	//zero the container used to hold the correct pass
	hipMemcpyToSymbol(HIP_SYMBOL(correctPass), &cpuCorrectPass, MAX_TOTAL, 0, hipMemcpyHostToDevice);

	//create and copy the charset to device
	hipMemcpyToSymbol(HIP_SYMBOL(cudaCharSet), &charSet, charSetLen, 0, hipMemcpyHostToDevice);

	bool finished = false;
	int ct = 0;
	do{
		hipMemcpyToSymbol(HIP_SYMBOL(cudaBrute), &currentBrute, MAX_BRUTE_LENGTH, 0, hipMemcpyHostToDevice);
		
		//run the kernel
		dim3 dimGrid(BLOCKS);
    		dim3 dimBlock(THREADS_PER_BLOCK);

		crack<<<dimGrid, dimBlock>>>(numThreads, charSetLen, wordLength, v1,v2,v3,v4);

		//get the "correct pass" and see if there really is one
		hipMemcpyFromSymbol(&cpuCorrectPass, HIP_SYMBOL(correctPass), MAX_TOTAL, 0, hipMemcpyDeviceToHost);

		if(cpuCorrectPass[0] != 0)
		{
			printf("\n\nFOUND: ");
			int k = 0;
			while(cpuCorrectPass[k] != 0)
			{
				printf("%c", cpuCorrectPass[k]);
				k++;
			}
			printf("\n");
			return 0;
		}
		
		finished = BruteIncrement(currentBrute, charSetLen, wordLength, numThreads * MD5_PER_KERNEL);

		checkCUDAError("general");
		
		if(ct % OUTPUT_INTERVAL == 0)
		{
			printf("STATUS: ");
			int k = 0;
			for(k = 0; k < wordLength; k++)
				printf("%c",charSet[currentBrute[k]]);
			printf("\n");
		}
		ct++;
		checkCUDAError("mehhhh");
	} while(!finished);

	return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(-1);
    }                         
}
