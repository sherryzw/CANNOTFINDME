#include "hip/hip_runtime.h"
//This is our CUDA thread
//d_a is the word list array
//maxidx is the maximum index in the array (if there are more threads than words)
//v1 through v4 are the uint values of the correct md5 hash

__device__ void IncrementBruteGPU(unsigned char* ourBrute, uint charSetLen, uint bruteLength, uint incrementBy)
{
	int i = 0;
	while(incrementBy > 0 && i < bruteLength)
	{
		int add = incrementBy + ourBrute[i];
		ourBrute[i] = add % charSetLen;
		incrementBy = add / charSetLen;
		i++;
	}
}

__global__ void crack(uint numThreads, uint charSetLen, uint bruteLength, uint v1, uint v2, uint v3, uint v4)
{
	//compute our index number
    uint idx = (blockIdx.x*blockDim.x + threadIdx.x);
	int totalLen = 0;
	int bruteStart = 0;

	unsigned char word[MAX_TOTAL];
	unsigned char ourBrute[MAX_BRUTE_LENGTH];

	int i = 0;

	for(i = 0; i < MAX_BRUTE_LENGTH; i++)
	{
		ourBrute[i] = cudaBrute[i];
	}
	
	i = 0;
	int ary_i = 0;
	unsigned char tmp = 0;
	while((tmp = cudaLeftSalt[ary_i]) != 0)
	{
		word[i] = tmp;
		i++; ary_i++;
	}
	bruteStart = i;
	i+= bruteLength;
	ary_i = 0;
	while((tmp = cudaRightSalt[ary_i]) != 0)
	{
		word[i] = tmp;
		i++; ary_i++;
	}
	totalLen = i;

	IncrementBruteGPU(ourBrute, charSetLen, bruteLength, idx);
	int timer = 0;
	for(timer = 0; timer < MD5_PER_KERNEL; timer++)
	{
		
		
		//Now, substitute the values into the string
		for(i = 0; i < bruteLength; i++)
		{
			word[i+bruteStart] = cudaCharSet[ourBrute[i]];
		}

		uint c1 = 0, c2 = 0, c3 = 0, c4 = 0;
		//get the md5 hash of the word
		md5_vfy(word,totalLen, &c1, &c2, &c3, &c4);
	
		//compare hash with correct hash
		if(c1 == v1 && c2 == v2 && c3 == v3 && c4 == v4)
		{
			//put the correct password in the first indexes of the array, right after the sentinal
			int j;
			for(j= 0; j < MAX_TOTAL; j++)
			{
				correctPass[j] = word[j];
			}
			correctPass[totalLen] = 0;
		}
		IncrementBruteGPU(ourBrute, charSetLen, bruteLength, numThreads);
	}
}

